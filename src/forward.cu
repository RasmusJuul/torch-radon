#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "utils.h"
#include "texture.h"
#include "parameter_classes.h"
#include "log.h"
#include "vectors.cuh"


template<bool parallel_beam, int channels, typename T>
__global__ void
radon_forward_kernel(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                     const VolumeCfg vol_cfg, const Projection2D proj_cfg) {

    // Calculate texture coordinates
    const int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;

    const int base = ray_id + proj_cfg.det_count * (angle_id + proj_cfg.n_angles * blockIdx.z);
    const int mem_pitch = proj_cfg.det_count * proj_cfg.n_angles * blockDim.z * gridDim.z;

    if (angle_id < proj_cfg.n_angles && ray_id < proj_cfg.det_count) {
        float accumulator[channels];

#pragma unroll
        for (int i = 0; i < channels; i++) accumulator[i] = 0.0f;

        // compute ray
        float sx, sy, ex, ey;
        if (parallel_beam) {
            sx = (ray_id - proj_cfg.det_count * 0.5f + 0.5f) * proj_cfg.det_spacing;
            sy = hypot(abs(proj_cfg.dx) + vol_cfg.width * 0.5f, abs(proj_cfg.dy) + vol_cfg.height * 0.5f);
            ex = sx;
            ey = -sy;
        } else {
            sy = proj_cfg.s_dist;
            sx = 0.0f;
            ey = -proj_cfg.d_dist;
            ex = (ray_id - proj_cfg.det_count * 0.5f + 0.5f) * proj_cfg.det_spacing;
        }

        // rotate ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        // start position rs and direction rd (in detector coordinate system)
        float rsx = sx * cs + sy * sn;
        float rsy = -sx * sn + sy * cs;
        float rdx = ex * cs + ey * sn - rsx;
        float rdy = -ex * sn + ey * cs - rsy;

        // convert coordinates to volume coordinate system
        const float vol_orig_x = proj_cfg.dx - 0.5f * vol_cfg.width * vol_cfg.spacing.x;
        const float vol_orig_y = proj_cfg.dy - 0.5f * vol_cfg.height * vol_cfg.spacing.y;
        rsx = (rsx - vol_orig_x) * vol_cfg.inv_spacing.x;
        rsy = (rsy - vol_orig_y) * vol_cfg.inv_spacing.y;
        rdx = rdx * vol_cfg.inv_spacing.x;
        rdy = rdy * vol_cfg.inv_spacing.y;


        // clip to volume (to reduce memory reads)
        float dx = rdx >= 0 ? max(rdx, 1e-6) : min(rdx, -1e-6);
        float dy = rdy >= 0 ? max(rdy, 1e-6) : min(rdy, -1e-6);

        const float alpha_x_m = (-rsx) / dx;
        const float alpha_x_p = (vol_cfg.width - rsx) / dx;
        const float alpha_y_m = (-rsy) / dy;
        const float alpha_y_p = (vol_cfg.height - rsy) / dy;
        const float alpha_s = max(min(alpha_x_p, alpha_x_m), min(alpha_y_p, alpha_y_m));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), max(alpha_y_p, alpha_y_m));

        // if ray volume intersection is empty exit
        if (alpha_s > alpha_e - 1e-6) {
#pragma unroll
            for (int b = 0; b < channels; b++) output[base + b * mem_pitch] = 0.0f;
            return;
        }

        rsx += rdx * alpha_s;
        rsy += rdy * alpha_s;
        rdx *= (alpha_e - alpha_s);
        rdy *= (alpha_e - alpha_s);

        const int n_steps = __float2int_rn(max(abs(rdx), abs(rdy)));
        const float vx = rdx / max(abs(rdx), abs(rdy));
        const float vy = rdy / max(abs(rdx), abs(rdy));
        const float n = hypot(vx * vol_cfg.spacing.x, vy * vol_cfg.spacing.y);

        float step;
        if (abs(rdy) >= abs(rdx)) {
            float y_increment = 0.5f - rsy + __float2int_rn(rsy);
            step = y_increment / vy;
            step += vy < 0;
        } else {
            float x_increment = 0.5f - rsx + __float2int_rn(rsx);
            step = x_increment / vx;
            step += vx < 0;
        }
        rsx += step * vx;
        rsy += step * vy;

#pragma unroll(4)
        for (int j = 0; j < n_steps; j++) {
            if (channels == 1) {
                accumulator[0] += tex2DLayered<float>(texture, rsx, rsy, blockIdx.z);
            } else {
                float4 read = tex2DLayered<float4>(texture, rsx, rsy, blockIdx.z);

                accumulator[0] += read.x;
                accumulator[1] += read.y;
                accumulator[2] += read.z;
                accumulator[3] += read.w;
            }
            rsx += vx;
            rsy += vy;
        }

#pragma unroll
        for (int b = 0; b < channels; b++) output[base + b * mem_pitch] = accumulator[b] * n;
    }
}

template<typename T>
void radon_forward_cuda(
        const T *x, const float *angles, T *y, TextureCache &tex_cache,
        const VolumeCfg &vol_cfg, const Projection2D &proj_cfg, const ExecCfg &exec_cfg,
        const int batch_size, const int device
) {
    constexpr bool is_float = std::is_same<T, float>::value;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = exec_cfg.get_channels(batch_size);

    LOG_DEBUG("Radon forward 2D. Height: " << vol_cfg.height << " width: " << vol_cfg.width << " channels: " << channels);
    LOG_DEBUG("Radon forward 2D. Det count: " << proj_cfg.det_count << " angles: " << proj_cfg.n_angles << " batch_size: " << batch_size);

    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get(
            {device, batch_size / channels, vol_cfg.height, vol_cfg.width, true, channels, precision});
    tex->put(x);

    // Invoke kernel
    const dim3 grid_dim = exec_cfg.get_grid_size(proj_cfg.det_count, proj_cfg.n_angles, batch_size / channels);
    const dim3 block_dim = exec_cfg.get_block_dim();

    LOG_DEBUG("Block Size x:" << block_dim.x << " y:" << block_dim.y << " z:" << block_dim.z);
    LOG_DEBUG("Grid Size x:" << grid_dim.x << " y:" << grid_dim.y << " z:" << grid_dim.z);

    if (proj_cfg.type == ProjectionType::FanBeam) {
        if (channels == 1) {
            radon_forward_kernel<false, 1> << < grid_dim, block_dim >> >
            ((float *) y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel<false, 4> << < grid_dim, block_dim >> >
                ((float *) y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel<false, 4> << < grid_dim, block_dim >> >
                ((__half *) y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    } else {
        if (channels == 1) {
            radon_forward_kernel<true, 1> << < grid_dim, block_dim >> >
            ((float *) y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel<true, 4> << < grid_dim, block_dim >> >
                ((float *) y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel<true, 4> << < grid_dim, block_dim >> >
                ((__half *) y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    }
}

template void
radon_forward_cuda<float>(const float *x, const float *angles, float *y, TextureCache &tex_cache,
                          const VolumeCfg &vol_cfg, const Projection2D &proj_cfg, const ExecCfg &exec_cfg,
                          const int batch_size, const int device);

template void radon_forward_cuda<ushort>(const ushort *x, const float *angles, ushort *y,
                                         TextureCache &tex_cache,
                                         const VolumeCfg &vol_cfg, const Projection2D &proj_cfg,
                                         const ExecCfg &exec_cfg,
                                         const int batch_size, const int device);


template<int channels, typename T>
__global__ void
radon_forward_kernel_3d(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                        const VolumeCfg vol_cfg, const Projection3D proj_cfg) {
    // Calculate sensor coordinates in pixels
    // TODO is there an "optimal" map from thread to coordinates that maximizes cache hits?
    // TODO check other permutations (combined with different block sizes)
    const int pu = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int pv = blockIdx.z * blockDim.z + threadIdx.z;

    const uint index = (angle_id * proj_cfg.det_count_v + pv) * proj_cfg.det_count_u + pu;
    const uint mem_pitch = proj_cfg.n_angles * proj_cfg.det_count_v * proj_cfg.det_count_u;

    if (angle_id < proj_cfg.n_angles && pu < proj_cfg.det_count_u && pv < proj_cfg.det_count_v) {
        // define accumulator
        float accumulator[channels];
#pragma unroll
        for (int i = 0; i < channels; i++) accumulator[i] = 0.0f;

        // compute ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        vec3 source = {0.0, -proj_cfg.s_dist, 0.0};
        float u = (pu - proj_cfg.det_count_u * 0.5f + 0.5f);
        float v = (pv - proj_cfg.det_count_v * 0.5f + 0.5f);
        vec3 dir = {u * proj_cfg.det_spacing_u, proj_cfg.s_dist + proj_cfg.d_dist, v * proj_cfg.det_spacing_v};

        // rotate start/end positions and add pitch * angle / (2*pi) to z
        source = rotxy_transz(source, sn, cs, proj_cfg.pitch * angle * 0.1591549f);
        dir = rotxy(dir, sn, cs);

        // convert coordinates to volume coordinate system
        source = proj_cfg.worldToVoxel * source;
        dir = rotate_scale(proj_cfg.worldToVoxel, dir);

        // Clip ray to cube to reduce the number of memory reads
        float dx = dir.x >= 0 ? max(dir.x, 1e-6) : min(dir.x, -1e-6);
        float dy = dir.y >= 0 ? max(dir.y, 1e-6) : min(dir.y, -1e-6);
        float dz = dir.z >= 0 ? max(dir.z, 1e-6) : min(dir.z, -1e-6);

        const float alpha_x_m = (-source.x) / dx;
        const float alpha_x_p = (vol_cfg.width - source.x) / dx;
        const float alpha_y_m = (-source.y) / dy;
        const float alpha_y_p = (vol_cfg.height - source.y) / dy;
        const float alpha_z_m = (-source.z) / dz;
        const float alpha_z_p = (vol_cfg.slices - source.z) / dz;

        const float alpha_s = max(min(alpha_x_p, alpha_x_m), max(min(alpha_y_p, alpha_y_m), min(alpha_z_p, alpha_z_m)));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), min(max(alpha_y_p, alpha_y_m), max(alpha_z_p, alpha_z_m)));

        if (alpha_s > alpha_e - 1e-6) {
#pragma unroll
            for (int b = 0; b < channels; b++) output[b * mem_pitch + index] = 0.0f;
            return;
        }

        source = source + dir * alpha_s;
        dir = dir * (alpha_e - alpha_s);

        // Compute number of steps
        const float f_n_steps = max(abs(dir.x), max(abs(dir.y), abs(dir.z)));
        const int n_steps = __float2uint_ru(f_n_steps);
        vec3 w = dir * __fdividef(1.0f, f_n_steps);
        const float n = norm(w * vol_cfg.spacing);

        // Do a small step to start near the middle of the first voxel (?)
        float step;
        if (abs(dir.y) >= abs(dir.x)) {
            float y_increment = 0.5f - source.y + __float2int_rn(source.y);
            step = y_increment / w.y;
            step += w.y < 0;
        } else {
            float x_increment = 0.5f - source.x + __float2int_rn(source.x);
            step = x_increment / w.x;
            step += w.x < 0;
        }
        source = source + step * w;

        // accumulate loop
#pragma unroll(4)
        for (int j = 0; j <= n_steps; j++) {
            if (channels == 1) {
                accumulator[0] += tex3D<float>(texture, source.x, source.y, source.z);
            } else {
                float4 read = tex3D<float4>(texture, source.x, source.y, source.z);
                accumulator[0] += read.x;
                accumulator[1] += read.y;
                accumulator[2] += read.z;
                accumulator[3] += read.w;
            }

            source += w;
        }

        // output
#pragma unroll
        for (int b = 0; b < channels; b++) {
            output[b * mem_pitch + index] = accumulator[b] * n;
        }
    }
}

template<typename T>
void radon_forward_cuda_3d(
        const T *x, const float *angles, T *y, TextureCache &tex_cache,
        const VolumeCfg &vol_cfg, Projection3D &proj_cfg, const ExecCfg &exec_cfg,
        const int batch_size, const int device
) {
    constexpr bool is_float = std::is_same<T, float>::value;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = exec_cfg.get_channels(batch_size);

    Texture *tex = tex_cache.get(
            {device, vol_cfg.slices, vol_cfg.height, vol_cfg.width, false, channels, precision});

    const dim3 grid_dim = exec_cfg.get_grid_size(proj_cfg.det_count_u, proj_cfg.n_angles, proj_cfg.det_count_v);
    const dim3 block_dim = exec_cfg.get_block_dim();

    proj_cfg.updateMatrices(vol_cfg);

    for (int i = 0; i < batch_size; i += channels) {
        T *local_y = &y[i * proj_cfg.det_count_u * proj_cfg.det_count_v * proj_cfg.n_angles];
        tex->put(&x[i * vol_cfg.slices * vol_cfg.height * vol_cfg.width]);

        // Invoke kernel
        if (channels == 1) {
            radon_forward_kernel_3d<1> << < grid_dim, block_dim >> >
            (local_y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel_3d<4> << < grid_dim, block_dim >> >
                (local_y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel_3d<4> << < grid_dim, block_dim >> >
                ((__half *) local_y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    }
}

template void
radon_forward_cuda_3d<float>(const float *x, const float *angles, float *y, TextureCache &tex_cache,
                             const VolumeCfg &vol_cfg, Projection3D &proj_cfg, const ExecCfg &exec_cfg,
                             const int batch_size, const int device);

template void radon_forward_cuda_3d<ushort>(const ushort *x, const float *angles, ushort *y,
                                            TextureCache &tex_cache,
                                            const VolumeCfg &vol_cfg, Projection3D &proj_cfg,
                                            const ExecCfg &exec_cfg,
                                            const int batch_size, const int device);


__global__ void radon_forward_kernel_3d_batch(float *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                                              const VolumeCfg vol_cfg, const Projection3D *__restrict__ proj_cfgs, int n_angles) {
    // Calculate sensor coordinates in pixels
    const int pu = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int pv = blockIdx.z * blockDim.z + threadIdx.z;

    const uint index = (angle_id * proj_cfgs[0].det_count_v + pv) * proj_cfgs[0].det_count_u + pu;

    if (angle_id < n_angles && pu < proj_cfgs[0].det_count_u && pv < proj_cfgs[0].det_count_v) {
        const Projection3D proj_cfg = proj_cfgs[angle_id];

        // define accumulator
        float accumulator = 0.0f;

        // compute ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        vec3 source = {0.0, -proj_cfg.s_dist, 0.0};
        float u = (pu - proj_cfg.det_count_u * 0.5f + 0.5f);
        float v = (pv - proj_cfg.det_count_v * 0.5f + 0.5f);
        vec3 dir = {u * proj_cfg.det_spacing_u, proj_cfg.s_dist + proj_cfg.d_dist, v * proj_cfg.det_spacing_v};

        // rotate start/end positions and add pitch * angle / (2*pi) to z
        source = rotxy_transz(source, sn, cs, proj_cfg.pitch * angle * 0.1591549f);
        dir = rotxy(dir, sn, cs);

        // convert coordinates to volume coordinate system
        source = proj_cfg.worldToVoxel * source;
        dir = rotate_scale(proj_cfg.worldToVoxel, dir);

        // Clip ray to cube to reduce the number of memory reads
        float dx = dir.x >= 0 ? max(dir.x, 1e-6) : min(dir.x, -1e-6);
        float dy = dir.y >= 0 ? max(dir.y, 1e-6) : min(dir.y, -1e-6);
        float dz = dir.z >= 0 ? max(dir.z, 1e-6) : min(dir.z, -1e-6);

        const float alpha_x_m = (-source.x) / dx;
        const float alpha_x_p = (vol_cfg.width - source.x) / dx;
        const float alpha_y_m = (-source.y) / dy;
        const float alpha_y_p = (vol_cfg.height - source.y) / dy;
        const float alpha_z_m = (-source.z) / dz;
        const float alpha_z_p = (vol_cfg.slices - source.z) / dz;

        const float alpha_s = max(min(alpha_x_p, alpha_x_m), max(min(alpha_y_p, alpha_y_m), min(alpha_z_p, alpha_z_m)));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), min(max(alpha_y_p, alpha_y_m), max(alpha_z_p, alpha_z_m)));

        if (alpha_s > alpha_e - 1e-6) {
            output[index] = 0.0f;
            return;
        }

        source = source + dir * alpha_s;
        dir = dir * (alpha_e - alpha_s);

        // Compute number of steps
        const float f_n_steps = max(abs(dir.x), max(abs(dir.y), abs(dir.z)));
        const int n_steps = __float2uint_ru(f_n_steps);
        vec3 w = dir * __fdividef(1.0f, f_n_steps);
        const float n = norm(w * vol_cfg.spacing);

        // Do a small step to start near the middle of the first voxel (?)
        float step;
        if (abs(dir.y) >= abs(dir.x)) {
            float y_increment = 0.5f - source.y + __float2int_rn(source.y);
            step = y_increment / w.y;
            step += w.y < 0;
        } else {
            float x_increment = 0.5f - source.x + __float2int_rn(source.x);
            step = x_increment / w.x;
            step += w.x < 0;
        }
        source = source + step * w;

        // accumulate loop
#pragma unroll(4)
        for (int j = 0; j <= n_steps; j++) {
            accumulator += tex3D<float>(texture, source.x, source.y, source.z);
            source += w;
        }

        output[index] = accumulator;
    }
}

void radon_forward_cuda_3d_batch(
        const float *x, const float *angles, float *y, TextureCache &tex_cache,
        const VolumeCfg &vol_cfg, std::vector <Projection3D> &proj_cfgs, const ExecCfg &exec_cfg,
        const int batch_size, const int device
) {
    Texture *tex = tex_cache.get({device, vol_cfg.slices, vol_cfg.height, vol_cfg.width, false, 1, PRECISION_FLOAT});

    const dim3 grid_dim = exec_cfg.get_grid_size(proj_cfgs[0].det_count_u, proj_cfgs.size(), proj_cfgs[0].det_count_v);
    const dim3 block_dim = exec_cfg.get_block_dim();

    for (auto &proj_cfg: proj_cfgs)
        proj_cfg.updateMatrices(vol_cfg);

    Projection3D *cuda_proj_cfgs = nullptr;
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMalloc((void **) &cuda_proj_cfgs, proj_cfgs.size() * sizeof(Projection3D)));
    checkCudaErrors(hipMemcpy(cuda_proj_cfgs, &proj_cfgs[0], proj_cfgs.size() * sizeof(Projection3D), hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

    for (int i = 0; i < batch_size; i++) {
        float *local_y = &y[i * proj_cfgs[0].det_count_u * proj_cfgs[0].det_count_v * int(proj_cfgs.size())];
        tex->put(&x[i * vol_cfg.slices * vol_cfg.height * vol_cfg.width]);
        checkCudaErrors(hipDeviceSynchronize());

        radon_forward_kernel_3d_batch<<< grid_dim, block_dim >>>(local_y, tex->texture, angles, vol_cfg, cuda_proj_cfgs, int(proj_cfgs.size()));
        checkCudaErrors(hipDeviceSynchronize());
    }

    checkCudaErrors(hipFree(cuda_proj_cfgs));
    checkCudaErrors(hipDeviceSynchronize());
}
